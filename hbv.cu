#include "hip/hip_runtime.h"
/**************
* HydroCuda model
*
* HBV hydrological model
* using cuda
***************/

#include <math.h>

__global__ void hbv_dynamic( float *q    ,
                             float *etr  ,
                             float *qin  ,
                             float *qlz  ,
                             float *qq   ,
                             float *qi   ,
                             float *qcf  ,
                             float *sp   ,
                             float *inet ,
                             float *qdr  ,
                             float *pp   ,
                             float *etp  ,
                             float *sm   ,
                             float *fc   ,
                             float *vlz  ,
                             float *vuz  ,
                             float *cflux,
                             float *icf  ,
                             float *beta ,
                             float *lp   ,
                             float *perc ,
                             float *uzl  ,
                             float *klz  ,
                             float *ki   ,
                             float *kq   )
{
  const int i = blockDim.x*blockDim.y*blockIdx.y*gridDim.x + blockDim.x*blockDim.y*blockIdx.x + blockDim.x*threadIdx.y + threadIdx.x;
  // In flows
  qin[i]  = max(pp[i] - icf, 0.f);
  etr[i]  = min(icf, etp[i]);
  // State variables
  sm[i]   = sm[i] + qin[i];
  qdr[i]  = max(sm[i] - fc[i], 0.f);
  sm[i]   = sm[i] - qdr[i];
  inet[i] = qin[i] - qdr[i];
  sp[i]   = pow(sm[i] / fc[i], beta[i]) * inet[i];
  sm[i]   = sm[i] - sp[i];
  // Evapotranspirations
  etp[i]  = etp[i] - etr[i];
  etr[i]  = min(min(sm[i] * etp[i] / (lp[i] * fc[i]), etp[i]), sm[i]);
  sm[i]   = sm[i] - etr[i];
  // Volume states
  vlz[i]  = vlz[i] + min(perc[i], qdr[i] + sp[i]);
  vuz[i]  = vuz[i] + max(0.f, qdr[i] + sp[i] - perc[i]);
  // Capillar flow
  qcf[i]  = cflux[i] * (fc[i] - sm[i]) / fc[i];
  sm[i]   = sm[i] + min(vuz[i], qcf[i]);
  vuz[i]  = max(vuz[i] - qcf[i], 0.f);
  // Quick and inter flow
  qq[i]   = max(kq[i] * (vuz[i] - uzl[i]), 0.f);
  qi[i]   = ki[i] * min(uzl[i], vuz[i]);
  // Base flow
  qlz[i]  = klz[i] * vlz[i];
  // Final flow
  q[i]    = qlz[i] + qi[i] + qq[i];
  
}

