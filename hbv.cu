
#include <hip/hip_runtime.h>
#include <math.h>
__global__ void hbv_dynamic( float *q, float *etr, float *qin, float *qlz, float *qq, float *qi, float *qcf, float *sp, float *inet, float *qdr, float *pp, float *etp, float *sm, float *fc, float *vlz, float *vuz, const float cflux, const float icf, const float beta, const float lp, const float perc, float *uzl, const float klz, const float ki, const float kq)
{
  const int i = blockDim.x*blockDim.y*blockIdx.y*gridDim.x + blockDim.x*blockDim.y*blockIdx.x + blockDim.x*threadIdx.y + threadIdx.x;
  
  qin[i] = max(pp[i] - icf, 0.f);
  etr[i] = min(icf, etp[i]);
  
  sm[i] = sm[i] + qin[i];
  qdr[i] = max(sm[i] - fc[i], 0.f);
  sm[i] = sm[i] - qdr[i];
  inet[i] = qin[i] - qdr[i];
  sp[i] = pow(sm[i] / fc[i],beta) * inet[i];
  sm[i] = sm[i] - sp[i];
  
  etp[i] = etp[i] - etr[i];
  etr[i] = min(min(sm[i] * etp[i] / (lp * fc[i]),etp[i]),sm[i]);
  // etr[i] = etp[i];
  // if (sm[i] < (lp * fc[i])) {
  //     etr[i] = sm[i] * etp[i] / (lp * fc[i]) ;
  //     }
  sm[i] = sm[i] - etr[i];
  
  vlz[i] = vlz[i] + min(perc,qdr[i] + sp[i]);
  vuz[i] = vuz[i] + max(0.f,qdr[i] + sp[i] - perc);
  
  qcf[i] = cflux * (fc[i] - sm[i]) / fc[i];
  sm[i] = sm[i] + min(vuz[i],qcf[i]);
  vuz[i] = max(vuz[i] - qcf[i],0.f);
  
  qq[i] = max(kq * (vuz[i] - uzl[i]),0.f);
  qi[i] = ki * min(uzl[i],vuz[i]);
  
  qlz[i] = klz * vlz[i];
  
  q[i] = qlz[i] + qi[i] + qq[i];
}

